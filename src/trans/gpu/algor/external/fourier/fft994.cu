
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
extern "C"
void
fft994_(hipfftDoubleComplex *data_h, int *INCp, \
        int *JUMPp, int *Np, int *LOTp, int *ISIGNp)
{
hipfftHandle plan;
hipfftDoubleComplex *data;
int INC = *INCp;
int JUMP = *JUMPp;
int N = *Np;
int LOT = *LOTp;
int ISIGN = *ISIGNp;
int RANK = 1;

int INEMBED[]={N};
int ISTRIDE = INC;
int IDIST = JUMP;
int ONEMBED[]={N/2+1};
int OSTRIDE = INC;
int ODIST = JUMP;

int NN[1] = {N};
/*
printf("%s %d \n","sizeof(cufftDoubleComplex)=",sizeof(cufftDoubleComplex));
printf("%s %d \n","INC=",INC);
printf("%s %d \n","JUMP=",JUMP);
printf("%s %d \n","N=",N);
printf("%s %d \n","LOT=",LOT);
printf("%s %d \n","ISIGN=",ISIGN);
printf("%s %d \n","sizeof(cufftDoubleComplex)*(N/2+1)*LOT=",sizeof(cufftDoubleComplex)*(N/2+1)*LOT);
*/

hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*(N/2+1)*LOT);
if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return;	
}

if (hipDeviceSynchronize() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}

hipMemcpy( data, data_h, sizeof(hipfftDoubleComplex)*(N/2+1)*LOT, hipMemcpyHostToDevice );

if( ISIGN== -1 ){
  /*
  if (cufftPlan1d(&plan, N, CUFFT_D2Z, LOT) != CUFFT_SUCCESS){
	  fprintf(stderr, "CUFFT error(DIR): Plan creation failed");
	  return;	
  }	
  */
  if(hipfftPlanMany(&plan, RANK, NN, INEMBED, \
        ISTRIDE, IDIST, ONEMBED, OSTRIDE, \
        ODIST, HIPFFT_D2Z, LOT ) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error(DIR): Plan creation failed");
        return;
  }
  /* Use the CUFFT plan to transform the signal in place. */
  if (hipfftExecD2Z(plan, (hipfftDoubleReal*)data, data) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error(DIR): ExecD2Z failed");
	return;	
  }
}
else if( ISIGN== 1){
  /*
  if (cufftPlan1d(&plan, N, CUFFT_Z2D, LOT) != CUFFT_SUCCESS){
	  fprintf(stderr, "CUFFT error(INV): Plan creation failed");
	  return;	
  }	
  */
  if(hipfftPlanMany(&plan, RANK, NN, ONEMBED, \
        OSTRIDE, ODIST, INEMBED, ISTRIDE, \
        IDIST, HIPFFT_Z2D, LOT ) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error(INV): Plan creation failed");
        return;
  }
  /* Use the CUFFT plan to transform the signal in place. */
  if (hipfftExecZ2D(plan, data, (hipfftDoubleReal*)data) != HIPFFT_SUCCESS){
	fprintf(stderr, "CUFFT error(INV): ExecZ2D failed");
	return;	
  }
}
else {
  abort();
}


if (hipDeviceSynchronize() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}

hipMemcpy( data_h, data, sizeof(hipfftDoubleComplex)*(N/2+1)*LOT, hipMemcpyDeviceToHost );
hipfftDestroy(plan);
hipFree(data);

if (hipDeviceSynchronize() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}

}
