//
// Wrapper for cublasDgemm function. 
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h" 


bool alreadyAllocated=false;

double **d_Aarray;
double **d_Barray;
double **d_Carray;

double **Aarray;
double **Barray;
double **Carray;

hipblasHandle_t handle;	

extern "C" void cublasDgemmBatched_wrapper (char transa, char transb, int m, int n,int k, double alpha, const double *A, int lda, int tda, const double *B, int ldb, int tdb, double beta, double *C, int ldc, int tdc, int batchCount)
{


  // printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n",m,n,k,batchCount);
    hipblasStatus_t stat;

 
  hipblasOperation_t op_t1=HIPBLAS_OP_N, op_t2=HIPBLAS_OP_N;

  if (transa=='T' || transa=='t')	
    op_t1=HIPBLAS_OP_T;

  if (transb=='T' || transb=='t')
    op_t2=HIPBLAS_OP_T;


  //double **Aarray = (double**) malloc(batchCount*sizeof(double*));
  //double **Barray = (double**) malloc(batchCount*sizeof(double*));
  //double **Carray = (double**) malloc(batchCount*sizeof(double*));


  if (!alreadyAllocated){

     stat = hipblasCreate(&handle);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        //return EXIT_FAILURE;
    }
    printf("cublascreate return code : %d\n",stat);

    hipError_t errcm1 = hipHostMalloc(&Aarray,batchCount*sizeof(double*), hipHostMallocDefault);
    hipError_t errcm2 = hipHostMalloc(&Barray,batchCount*sizeof(double*), hipHostMallocDefault);
    hipError_t errcm3 = hipHostMalloc(&Carray,batchCount*sizeof(double*), hipHostMallocDefault);
        
    hipError_t errcm4 = hipMalloc(&d_Aarray,batchCount*sizeof(double*));
    hipError_t errcm5 = hipMalloc(&d_Barray,batchCount*sizeof(double*));
    hipError_t errcm6 = hipMalloc(&d_Carray,batchCount*sizeof(double*));
 
    printf("switched alreadyAllocated to true\n");
    printf("Allocation statuses : %d %d %d %d %d %d\n", errcm1, errcm2, errcm3, errcm4, errcm5, errcm6 );
    alreadyAllocated=true;
  }

  int i;
  for(i=0;i<batchCount;i++){
    Aarray[i]=(double*) &(A[i*lda*tda]);
    Barray[i]=(double*) &(B[i*ldb*tdb]);
    Carray[i]=(double*) &(C[i*ldc*tdc]);
  }

  hipError_t err1 = hipMemcpy(d_Aarray,Aarray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipError_t err2 = hipMemcpy(d_Barray,Barray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipError_t err3 = hipMemcpy(d_Carray,Carray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  printf("made it to the call to DgemmBatched ... are we already allocated? %d and err codes : %d %d %d \n",alreadyAllocated, err1, err2, err3);
  printf("batchCount etc : %d \n%d %d %d \n%d %d\n%d %d\n%d %d\n",batchCount, m,n,k, lda, tda, ldb,tdb, ldc,tdc);

  hipblasDgemmBatched(handle,op_t1,op_t2,m,n,k,&alpha,(const double**) d_Aarray,lda, (const double**) d_Barray,ldb,&beta,(double**) d_Carray,ldc,batchCount);

  //printf("after dgemm\n");
  hipDeviceSynchronize();
  
  //cudaFree(Aarray);
  //cudaFree(Barray);
  //cudaFree(Carray);
  
  //cudaFree(d_Aarray);
  //cudaFree(d_Barray);
  //cudaFree(d_Carray);
  //cublasDestroy(handle);
  
  
}

extern "C" void cublasDgemmBatched_finalize ()
{



  if (alreadyAllocated){
  
    hipFree(Aarray);
    hipFree(Barray);
    hipFree(Carray);
    
    hipFree(d_Aarray);
    hipFree(d_Barray);
    hipFree(d_Carray);
    hipblasDestroy(handle);

  }
  
}
